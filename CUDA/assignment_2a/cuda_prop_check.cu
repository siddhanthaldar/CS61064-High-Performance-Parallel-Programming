#include <stdio.h>
#include <hip/hip_runtime.h>


void  printDevProp(hipDeviceProp_t  devp)
{
	printf("No. of  multiprocessors: %d\n", devp.multiProcessorCount); //24
	printf("Size of warp %d\n", devp.warpSize ); //32
	printf("Max threads per block %d\n", devp.maxThreadsPerBlock);
	return;
}

int  main()
{
	int  devCount;
	hipGetDeviceCount(& devCount);
	for (int i = 0; i < devCount; ++i)
	{
		hipDeviceProp_t  devp;
		hipGetDeviceProperties(&devp ,i);
		printDevProp(devp);
	}
	return  0;
}

