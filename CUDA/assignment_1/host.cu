#include "hip/hip_runtime.h"
#include "headers.h"
/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 4*2*2*32*32;
    size_t size = numElements * sizeof(float);
    printf("Number of elements = %d\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_D = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_E = (float *)malloc(size);


    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    // Every function with a "cuda" prefix has a error code returned which can be used to track error
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Process Kernel 1 
    dim3 grid1(4,2,2);
    dim3 block1(32,32,1);
    process_kernel1<<<grid1,block1>>>(d_A,d_B,d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if ((sin(h_A[i]) + cos(h_B[i]) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    // for(int j=0; j<3;j++)
    // {
    //     printf("%f\t%f\n%f\n",h_A[j],h_B[j],h_C[j]);
    // }

    printf(" Completed Process 1\n");

    // Process Kernel 2 
    dim3 grid2(2,8,1);
    dim3 block2(8,8,16);
    
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_C, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    process_kernel2<<<grid2,block2>>>(d_A,d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_D, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if ((log(h_C[i]) - h_D[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

	// for(int j=0; j<3;j++)
 //    {
 //        printf("%f\n",h_D[j]);
 //    }
    printf(" Completed Process 2\n");    

    // Process Kernel 3 
    dim3 grid3(16,1,1);
    dim3 block3(128,8,1);
    
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_D, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    process_kernel3<<<grid3,block3>>>(d_A,d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_E, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if ((sqrt(h_D[i]) - h_E[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

	// for(int j=0; j<3;j++)
 //    {
 //        printf("%f\n",h_E[j]);
 //    }
    printf(" Completed Process 3\n");    



    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);
    free(h_E);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

